#include "hip/hip_runtime.h"

#include <cmath>
#include "HeldKarp.cuh"




/* Returns the index of the subset in the memo array.
 *
 * Parameters:
 *      set - the set of numbers to get an index for; the source point should
 *            be 0
 *      size - the number of points in the whole TSP algorithm
 */
__device__
int cudaGetSetIndex(Set set, int size) {
    /*!
     * Unfortunately it is hard to explain how this arithmetic works.
     * Basically, we know that in a set S with |S| = n, there are 2^n
     * subsets.  Thus, there are 2^(n - 1) subsets whose first number is
     * fixed.  However, if we put the constraint that it is only considered
     * a subset if it is sorted, then we find that there are always
     * 2 ^ (n - m) subsets when we fix the first two digits (remember the
     * subsets are not repeated, so these first two digits are smaller than
     * any other value in the set).  This was found purely by looking for
     * a pattern.  Using this, we can find a unique index for any subset in
     * O(log(n)) time.
     */

    // Sort the list so we can find its index.
    set.sort();

    // We will continually add to the returned index
    int memoIndex = 0;

    // Remember the lowest value we havent seen.  We start at 1 because the
    //    smallest subset that makes sense in this problem has two elements.
    //    Thus, every set must have at least one value 1 or greater.
    int lowest = 1;

    // This is the index in the set we are currently iterating over.  We start
    //    at 1 because the first element will always be the same (because we
    //    have a fixed first point in the problem)
    int setIndex = 1;

    while (1) {
        // Add in values for every subset of this subset we skip over
        for (; lowest < set[setIndex]; lowest++)
            memoIndex += powf(2, size - lowest - 1);

        // Increment the lowest value so that we don't double-check it.
        lowest++;
        setIndex++;

        // Break if we have seen every index
        if (set.nValues == setIndex)
            return memoIndex;

        // Increment the memo index because of a zero case that occurs if the
        //    next iteration is what was guessed.
        memoIndex++;
    }
}









/**
 * Gets all of the distances between any two points
 * 
 * 
 * points - List of x, y coordinates of points to find distances between.
 * nPoints - Number of points
 * distances - Array of distances between pairs of points.
 */
__global__
void cudaGetDistances(Point2D *points, int nPoints, float *distances) {

    // Get the index of the thread so we only iterate part of the data.
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Variables when filling in the distances array
    int row, col;


    while (tid < (nPoints * nPoints)) {
        // The row and column can be determined from mod and division
        row = tid / nPoints;
        col = tid % nPoints;

        // Get Euclidean distance and put it into the array.
        distances[tid] = points[row].distanceTo(points[col]);

        // Advance thread index.
        tid += blockDim.x * gridDim.x;
    }

}




void cudaCallGetDistances(int nBlocks,
                          int threadsPerBlock,
                          Point2D *points,
                          int nPoints,
                          float *distances) {

    // Number of bytes of shared memory
    int shmem = 0;

    // Fill in all of the distances between two points.
    cudaGetDistances<<<nBlocks, threadsPerBlock, shmem>>>(points, nPoints, distances);

}








/**
 * Gets the first rows of the memoization array so the rest of the algorithm can
 * run.  These are the rows for every set that has only two points (the first
 * is always the source point).
 * 
 * 
 * memoArray - The memoization array whose first rows will be initialized
 * points - The (x, y) coordinates of points that will be memoized.
 * nPoints - The number of points
 * distances - Distances between every pair of two points.
 */
__global__
void cudaInitializeMemoArray(HeldKarpMemoArray memoArray,
                             Point2D *points,
                             int nPoints,
                             float *distances) {

    // Get the index of the thread so we only iterate part of the data.
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // We don't care about the 0 to 0 case, so we will skip right away if
    // tid is 0.
    tid = (tid == 0 ? blockDim.x * gridDim.x : tid);

    while (tid < nPoints) {
        // Create a length two subset with the source as the first point
        int setPoints[2] = { 0, tid };

        // Memoize the "shortest distance" as the distance between these points.
        memoArray[cudaGetSetIndex(Set(setPoints, 2), nPoints)].updateRow(tid, distances[tid], 0);

        // Advance thread index.
        tid += blockDim.x * gridDim.x;
    }

}





void cudaCallInitializeMemoArray(int nBlocks,
                                 int threadsPerBlock,
                                 HeldKarpMemoArray memoArray,
                                 Point2D *points,
                                 int nPoints,
                                 float *distances) {

    // Number of bytes of shared memory
    int shmem = 0;

    // Initialize the memo array withs subsets of length 2
    cudaInitializeMemoArray<<<nBlocks, threadsPerBlock, shmem>>>
        (memoArray, points, nPoints, distances);

}





/**
 * Calculates the distance of the path through all points ending in any two
 * points.  The shortest of these will then be found in a different kernel.
 * 
 * 
 * set - The set of points to find the paths between.
 * memoArray - The memoization array from which to draw information.
 * distances - Distances between every pair of two points.
 * nPoints - Number of points.
 * mins - Array of distance/previous pairs that is filled by this function
 *        and left for another kernel to find the minimum of.
 */
__global__
void cudaHeldKarpKernel(Set set, 
                        HeldKarpMemoArray memoArray,
                        float *distances,
                        int nPoints,
                        HeldKarpMemo *mins) {

    // Get the index of the thread so we only iterate part of the data.
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Variables required.
    int m, k;
    
    // Finding every combination of m and k in one motion.  We will store the
    // distance and prev for each of these combinations, then another kernel
    // will find the minimum of all of these values for each k.  For more info
    // on m and k, refer to HeldKarp.cc  We will treat the mins array as an
    // array with set.nValues rows and set.nValues - 1 columns.
    while (tid < (set.nValues * (set.nValues - 1))) {
        // Get k and m from the tid
        
        k = tid / (set.nValues - 1); // Index of value subtracting from set
        m = tid % (set.nValues - 1); // Value asserting as last in set
        
        // We never want 0 to be last, and last can't also be removed from set
        if (m != k && set[m] != 0) {
			
            // Remove k from set to look at shortest path ending in m, k
            Set newSet = set - set[k];
            
            // Store the distance and prev in mins to get the min later.
            HeldKarpMemoRow memo = memoArray[cudaGetSetIndex(newSet, nPoints)];
            
            if ((memo[newSet[m]].dist + distances[newSet[m] + set[k]] < mins[k].dist) ||
                        (mins[k].dist == 0)) {
                                        
                mins[k].dist = memo[newSet[m]].dist + distances[newSet[m] + set[k]];
                mins[k].prev = newSet[m];
				
            }
            
            
        }
        
        // Advance thread index.
        
        tid += blockDim.x * gridDim.x;
    }
    
    __syncthreads();
    
     for (int k = 0; k < set.nValues; k++) {
         memoArray[cudaGetSetIndex(set, nPoints)].updateRow(set[k], mins[k].dist, mins[k].prev);
     }
    
}




void cudaCallHeldKarpKernel(int nBlocks,
                            int threadsPerBlock,
                            Set set,
                            HeldKarpMemoArray memoArray,
                            float *distances,
                            int nPoints,
                            HeldKarpMemo *mins) {

    cudaHeldKarpKernel<<<nBlocks, threadsPerBlock>>> \
        (set, memoArray, distances, nPoints, mins);

}

